#include "hip/hip_runtime.h"
#include "gpu.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdlib>
#include <iostream>

__device__ uint64_t getDiff(uint32_t* buf1, uint32_t* buf2, int pos) {
	int32_t red1 = (uint8_t)(buf1[pos] >> 16);
	int32_t green1 = (uint8_t)(buf1[pos] >> 8);
	int32_t blue1 = (uint8_t)(buf1[pos]);
	int32_t red2 = (uint8_t)(buf2[pos] >> 16);
	int32_t green2 = (uint8_t)(buf2[pos] >> 8);
	int32_t blue2 = (uint8_t)(buf2[pos]);

	int64_t redDiff = abs(red1 - red2);
	int64_t greenDiff = abs(green1 - green2);
	int64_t blueDiff = abs(blue1 - blue2);

	return  redDiff + greenDiff + blueDiff;
}

__global__ void compare_kernel(uint32_t* buf1, uint32_t* buf2, uint64_t* sum, int w, int h, uint64_t sampleGrid[100] = nullptr) {
	const int xPix = blockDim.x * blockIdx.x + threadIdx.x;
	const int yPix = blockDim.y * blockIdx.y + threadIdx.y;

	unsigned int pos = w * yPix + xPix;

	uint64_t diff = getDiff(buf1, buf2, pos);
	atomicAdd(sum, diff);
	if (sampleGrid != nullptr) {
		atomicAdd(&sampleGrid[xPix * 10 / w + 10 * (yPix * 10 / (h+1))], diff);
	}
}

uint64_t gpuCompare(uint32_t* input, uint32_t* output, int size, int w, int h, uint64_t sampleGrid[100])
{
	const dim3 blocksPerGrid((w / TILE_WIDTH), (h / TILE_HEIGHT));
	const dim3 threadsPerBlock(TILE_WIDTH, TILE_HEIGHT);

	uint64_t* sum;
	hipMallocManaged((void**)&sum, sizeof(uint64_t));

	uint32_t* in = nullptr;
	uint32_t* out = nullptr;

	hipMallocManaged((void**)&in, size);
	hipMallocManaged((void**)&out, size);

	hipMemcpy(in, input, size, hipMemcpyHostToDevice);
	hipMemcpy(out, output, size, hipMemcpyHostToDevice);

	if (sampleGrid != nullptr) {
		uint64_t* grid = nullptr;
		int gridSize = sizeof(uint64_t) * 100;
		hipMallocManaged((void**)&grid, gridSize);
		hipMemcpy(grid, sampleGrid, gridSize, hipMemcpyHostToDevice);

		compare_kernel <<<blocksPerGrid, threadsPerBlock>>> (in, out, sum, w, h, grid);

		hipMemcpy(sampleGrid, grid, gridSize, hipMemcpyDeviceToHost);
		hipFree(grid);
	} else {
		compare_kernel <<<blocksPerGrid, threadsPerBlock>>> (in, out, sum, w, h);
	}

	hipDeviceSynchronize();

	uint64_t result = *sum;

	hipFree(sum);
	hipFree(in);
	hipFree(out);

	return result;
}

__global__ void difference_kernel(uint32_t* buf1, uint32_t* buf2, uint32_t* buf3, int w) {
	const int xPix = blockDim.x * blockIdx.x + threadIdx.x;
	const int yPix = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int pos = w * yPix + xPix;

	buf3[pos] = getDiff(buf1, buf2, pos);
}

void renderDifference(uint32_t* input, uint32_t* output, uint32_t* difference, int size, int w, int h)
{
	const dim3 blocksPerGrid((w / TILE_WIDTH), (h / TILE_HEIGHT));
	const dim3 threadsPerBlock(TILE_WIDTH, TILE_HEIGHT);

	uint32_t* in = nullptr;
	uint32_t* out = nullptr;
	uint32_t* diff = nullptr;

	hipMallocManaged((void**)&in, size);
	hipMallocManaged((void**)&out, size);
	hipMallocManaged((void**)&diff, size);

	hipMemcpy(in, input, size, hipMemcpyHostToDevice);
	hipMemcpy(out, output, size, hipMemcpyHostToDevice);
	hipMemcpy(diff, difference, size, hipMemcpyHostToDevice);

	difference_kernel <<<blocksPerGrid, threadsPerBlock>>> (in, out, diff, w);

	hipDeviceSynchronize();

	hipMemcpy(difference, diff, size, hipMemcpyDeviceToHost);

	hipFree(in);
	hipFree(out);
	hipFree(diff);
}
